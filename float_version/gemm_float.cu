#include "hip/hip_runtime.h"
#include "gemm_float.cuh"
struct Matrix
{
	int width;
	int height;
	float *elements;
};

__device__ float getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}

__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C, Matrix *D)
{
	float Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = 0; i < A->width; ++i)
	{
		Cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	Cvalue += getElement(D, row, col);
	setElement(C, row, col, Cvalue);
}

float *gemm(const int input_width, const int input_height, const int weight_width, const int weight_height,
			float *weight, float *bias, float *input)
{
	Matrix *A, *B, *C, *D;

	hipMallocManaged((void **)&A, sizeof(Matrix));
	hipMallocManaged((void **)&B, sizeof(Matrix));
	hipMallocManaged((void **)&C, sizeof(Matrix));
	hipMallocManaged((void **)&D, sizeof(Matrix));

	A->width = input_width;
	A->height = input_height;
	B->width = weight_width;
	B->height = weight_height;
	C->width = B->width;
	C->height = A->height;
	D->width = B->width;
	D->height = A->height;
	A->elements = input;
	B->elements = weight;
	D->elements = bias;

	hipMallocManaged((void **)&C->elements, C->width * C->height * sizeof(float));

	dim3 blockSize(32, 32);
	dim3 gridSize((C->width + blockSize.x - 1) / blockSize.x,
				  (C->height + blockSize.y - 1) / blockSize.y);

	struct timeval t1, t2;
	gettimeofday(&t1, NULL);
	float timeuse;

	matMulKernel<<<gridSize, blockSize>>>(A, B, C, D);

	// hipDeviceSynchronize();

	gettimeofday(&t2, NULL);
	timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec) / 1000000.0;
	printf("Use Time:%fs\n", timeuse);

	return C->elements;
}

int test_gemm_main()
{
	float *test_inputd;
	float test_input[1 * 2];
	float *test_weightd;
	float test_weight[2 * 3];
	float *test_biasd;
	float test_bias[3];
	float *test_outputd;
	float test_output[3];

	for (int i = 0; i < 2; i++)
	{
		test_input[i] = float(i);
		printf("input %f\n", test_input[i]);
	}
	for (int i = 0; i < 2 * 3; i++)
	{
		test_weight[i] = float(i);
		printf("weight %f\n", test_weight[i]);
	}
	for (int i = 0; i < 3; i++)
	{
		test_bias[i] = float(1);
		printf("bias %f\n", test_bias[i]);
	}

	hipMalloc(&test_inputd, sizeof(float) * 2);
	hipMemcpy(test_inputd, test_input, sizeof(float) * 2, hipMemcpyHostToDevice);
	hipMalloc(&test_weightd, sizeof(float) * 2 * 3);
	hipMemcpy(test_weightd, test_weight, sizeof(float) * 2 * 3, hipMemcpyHostToDevice);
	hipMalloc(&test_biasd, sizeof(float) * 3);
	hipMemcpy(test_biasd, test_bias, sizeof(float) * 3, hipMemcpyHostToDevice);

	test_outputd = gemm(2, 1, 3, 2, test_weightd, test_biasd, test_inputd);

	hipMemcpy(test_output, test_outputd, sizeof(float) * 3, hipMemcpyDeviceToHost);
	for (int i = 0; i < 3; i++)
	{
		printf("%f\n", test_output[i]);
	}
	return 0;
}

// int test_gemm_main()
// {
// 	int width = w;
// 	int height = w;

// 	Matrix *A, *B, *C;

// 	hipMallocManaged((void**)&A, sizeof(Matrix));
// 	hipMallocManaged((void**)&B, sizeof(Matrix));
// 	hipMallocManaged((void**)&C, sizeof(Matrix));

// 	int nBytes = width * height * sizeof(float);

// 	hipMallocManaged((void**)&A->elements, nBytes);
// 	hipMallocManaged((void**)&B->elements, nBytes);
// 	hipMallocManaged((void**)&C->elements, nBytes);

// 	A->height = height;
// 	A->width = width;
// 	B->height = height;
// 	B->width = width;
// 	C->height = height;
// 	C->width = width;

// 	for (int i = 0; i < width * height; ++i)
// 	{
// 		A->elements[i] = 1.0;
// 		B->elements[i] = 2.0;
// 	}

// 	dim3 blockSize(32, 32);
// 	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
// 		(height + blockSize.y - 1) / blockSize.y);

// 	struct timeval t1,t2;
// 	gettimeofday(&t1,NULL);
// 	float timeuse;

// 	matMulKernel << < gridSize, blockSize >> >(A, B, C);

// 	hipDeviceSynchronize();

// 	gettimeofday(&t2,NULL);
// 	timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec)/1000000.0;
// 	printf("Use Time:%fs\n", timeuse);

// 	return 0;
// }