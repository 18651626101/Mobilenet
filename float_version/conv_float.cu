#include "hip/hip_runtime.h"
#include "conv_float.cuh"
#define BLOCKSIZE 8
using namespace std;

/**
 * the output shape: (out_channel, out_width, out_width)
 * parallel: (out_width, out_width) with maximum #thread per block
 * with fusion of conv and clip
*/

__global__ void convKernel(float *input,float *filter,float *output, float* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, int next_padding, bool clip){
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    int input_row = output_row*stride;
    int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;
    out_width += next_padding * 2;

    int fnum = blockIdx.z;
    float tmp = 0.0;
    for(int d=0;d<input_depth;d++)
    for(int r=0;r<filter_width;r++)
    for(int c=0;c<filter_width;c++)
        tmp += input[d*input_width*input_width + (input_row+r)*input_width + input_col+c]*filter[fnum*input_depth*filter_width*filter_width + d*filter_width*filter_width + r*filter_width + c];
    
    tmp += bias[fnum];
    if(clip){
        if(tmp < 0) tmp = 0.0;
        else if(tmp > 6) tmp = 6.0;
    }
    output[fnum*out_width*out_width + (output_row+next_padding)*out_width+output_col+next_padding] = tmp;
}

__global__ void convGroupKernel(float *input,float *filter,float *output, float* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, int next_padding, bool clip){
    int layer_idx = blockIdx.z;
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    int input_row = output_row*stride;
    int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;
    out_width += next_padding * 2;

    
        float tmp = 0.0;
        for(int r=0;r<filter_width;r++)
        for(int c=0;c<filter_width;c++){
            tmp += input[layer_idx*input_width*input_width + (input_row+r)*input_width + input_col+c]*filter[layer_idx*filter_width*filter_width + r*filter_width + c];
        }
        
        tmp += bias[layer_idx];
        if(clip){
            if(tmp < 0) tmp = 0.0;
            else if(tmp > 6) tmp = 6.0;
        }
        output[layer_idx*out_width*out_width + (output_row+next_padding)*out_width+output_col+next_padding] = tmp;
    
}


/* the float*  are assumed to point to cuda mem. */
void conv(const int input_depth, const int input_width, 
    const int filter_num, const int out_width_,
    const int filter_width,
    const int padding, const int stride, const int dilation,
    float* filter, float* bias,
    float* &input, float* &output, int next_padding, const bool clip){
    // printf("========== conv_malloc::begin conv ==========\n");
    float *img_cuda,*filter_cuda,*output_cuda,*bias_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    int out_width=out_width_+next_padding*2;
    size_t outsize = sizeof(float)*filter_num*out_width*out_width;
    

    // hipMalloc(&output_cuda,outsize);
    output_cuda = output;
    hipMemset(output_cuda, 0, outsize);
    filter_cuda = filter;
    bias_cuda=bias;
    img_cuda = input;

    int g=(out_width_+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 threads(BLOCKSIZE, BLOCKSIZE);
    dim3 grid(g, g, filter_num);

    convKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width_, next_padding, clip);

    
    input = output_cuda;
    // printf("========== conv_v1::end conv ==========\n");
    
}

void conv_group(const int input_depth, const int input_width, 
    const int filter_num, const int out_width_,
    const int filter_width,
    const int padding, const int stride, const int dilation,
    float* filter, float* bias,
    float* &input, float* &output,int next_padding, const bool clip){
    // printf("========== conv_v1::begin conv ==========\n");
    float *img_cuda,*filter_cuda,*output_cuda,*bias_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    int out_width=out_width_+next_padding*2;
    size_t outsize = sizeof(float)*filter_num*out_width*out_width;

    // hipMalloc(&output_cuda,outsize);
    output_cuda = output;
    hipMemset(output_cuda,0, outsize);
    filter_cuda = filter;
    bias_cuda=bias;

    img_cuda = input;    

    dim3 threads(1, 1);
    dim3 grid(out_width_, out_width_, input_depth);

    convGroupKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width_,next_padding,clip);

    input = output;

    // printf("========== conv_v1::end conv ==========\n");
    
}

float* pad(const int input_depth, const int input_width, const int padding, float* input){
    float *img_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    hipMalloc(&img_cuda,sizeof(float)*input_depth*in_width*in_width);
    hipMemset(img_cuda, 0, sizeof(float)*input_depth*in_width*in_width);
    for(int mapid=0;mapid<input_depth;mapid++)
    for(int line=0;line<input_width;line++){
        hipMemcpy(&img_cuda[mapid*in_width*in_width+(padding+line)*in_width+padding], &input[mapid*input_width*input_width+line*input_width], sizeof(float)*input_width, hipMemcpyHostToDevice);
    }
    return img_cuda;
}

// int test_conv1_main()
// {
//     float* filter=new float[2*3*3*3];
//     float* image=new float[3*5*5];
//     float* out= new float[2*25];
//     float* bias = new float[2];
//     for(int i=0;i<2*3*3*3;i++)filter[i]=1;
//     for(int i=0;i<3*5*5;i++)image[i]=1;
//     bias[0] = 100; bias[1] = 10000;
//     float* filter_cu, *image_cu, *out_cu, *bias_cu;
//     struct timeval t0;
//     gettimeofday(&t0, NULL);
//     hipMalloc(&filter_cu, 2*3*3*3*sizeof(float));
//     hipMemcpy(filter_cu, filter,2*3*3*3*sizeof(float), hipMemcpyHostToDevice);
//     hipMalloc(&image_cu, 3*5*5*sizeof(float));
//     hipMemcpy(image_cu, image, 3*5*5*sizeof(float), hipMemcpyHostToDevice);
//     hipMalloc(&bias_cu, 2*sizeof(float));
//     hipMemcpy(bias_cu, bias,2*sizeof(float),hipMemcpyHostToDevice);
//     struct timeval t1, t2;
//     gettimeofday(&t1, NULL);
// 	float timeuse;
//     out_cu = conv(3,5,2,5,3,1,1,0,filter_cu,bias_cu,image_cu, false);
//     gettimeofday(&t2, NULL);
// 	timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec) / 1000000.0;
// 	printf("Conv Use Time:%fs\n", timeuse);
//     timeuse = t1.tv_sec - t0.tv_sec + (t1.tv_usec - t0.tv_usec) / 1000000.0;
// 	printf("Malloc Use Time:%fs\n", timeuse);
//     hipMemcpy(out, out_cu, 50*sizeof(float), hipMemcpyDeviceToHost);
//     for(int i=0;i<50;i++)printf("%f\n",out[i]);
//     return 0;
// }
// int main(){test_conv1_main(); return 0;}