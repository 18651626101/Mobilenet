#include "hip/hip_runtime.h"
#include "add.cuh"
#define BLOCKSIZE 32
using namespace std;

__global__ void AddKernel(const int size, float *inputA, float *inputB)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    inputA[k] += inputB[k];

}

void add(const int channel, const int shape, float *inputA, float *inputB)
{
    // printf("========== add:: begin add ==========\n");
    int size = channel*shape*shape;
    int grid_sz = size / BLOCKSIZE;
    if (size % BLOCKSIZE)
        grid_sz++;

    //invoke function on device
    dim3 dimGrid(grid_sz);
    dim3 dimBlock(BLOCKSIZE);
    AddKernel<<<dimGrid, dimBlock>>>(size, (float *)inputA, (float *)inputB);

    //return result
    hipFree(inputB);
    // printf("========== add:: end add ==========\n");
}

// int main()
int test_add_main()
{
    float *test_inputda;
    float test_inputa[3 * 2 * 2];
    float *test_inputdb;
    float test_inputb[3 * 2 * 2];
    // float *test_outputd;
    float test_output[3 * 2 * 2];

    for (int i = 0; i < 12; i++)
    {
        test_inputa[i] = float(i);
        test_inputb[i] = float(i);
        printf("%f %f\n", test_inputa[i], test_inputb[i]);
    }

    hipMalloc(&test_inputda, sizeof(float) * 3 * 2 * 2);
    hipMemcpy(test_inputda, test_inputa, sizeof(float) * 3 * 2 * 2, hipMemcpyHostToDevice);
    hipMalloc(&test_inputdb, sizeof(float) * 3 * 2 * 2);
    hipMemcpy(test_inputdb, test_inputb, sizeof(float) * 3 * 2 * 2, hipMemcpyHostToDevice);   
    
    add(3, 2, test_inputda, test_inputdb);

    hipMemcpy(test_output, test_inputda, sizeof(float) * 3*2*2, hipMemcpyDeviceToHost);
    for (int i = 0; i < 12; i++)
    {
        printf("%f\n", test_output[i]);
    }
    return 0;
}