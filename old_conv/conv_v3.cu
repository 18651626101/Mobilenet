#include "hip/hip_runtime.h"
#include "conv_v2.cuh"
#define BLOCKSIZE 32
using namespace std;

/**
 * the output shape: (out_channel, out_width, out_width)
 * parallel: (out_width, out_width) with maximum #thread per block
 * with fusion of conv and clip
*/

__global__ void convKernel(double *input,double *filter,double *output, double* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, bool clip){
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    int input_row = output_row*stride;
    int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;

    int fnum = blockIdx.z;
    double tmp = 0.0;
    for(int d=0;d<input_depth;d++)
    for(int r=0;r<filter_width;r++)
    for(int c=0;c<filter_width;c++)
        tmp += input[d*input_width*input_width + (input_row+r)*input_width + input_col+c]*filter[fnum*input_depth*filter_width*filter_width + d*filter_width*filter_width + r*filter_width + c];
    
    tmp += bias[fnum];
    if(clip){
        if(tmp < 0) tmp = 0.0;
        else if(tmp > 6) tmp = 6.0;
    }
    output[fnum*out_width*out_width + output_row*out_width+output_col] = tmp;
}

__global__ void convGroupKernel(double *input,double *filter,double *output, double* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, bool clip){
    int layer_idx = blockIdx.z;
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    int input_row = output_row*stride;
    int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;

    
        double tmp = 0.0;
        for(int r=0;r<filter_width;r++)
        for(int c=0;c<filter_width;c++){
            tmp += input[layer_idx*input_width*input_width + (input_row+r)*input_width + input_col+c]*filter[layer_idx*filter_width*filter_width + r*filter_width + c];
        }
        
        tmp += bias[layer_idx];
        if(clip){
            if(tmp < 0) tmp = 0.0;
            else if(tmp > 6) tmp = 6.0;
        }
        output[layer_idx*out_width*out_width + output_row*out_width+output_col] = tmp;
    
}


/* the double*  are assumed to point to cuda mem. */
double* conv(const int input_depth, const int input_width, 
    const int filter_num, const int out_width_,
    const int filter_width,
    const int padding, const int stride, const int dilation,
    double* filter, double* bias,
    double* input, const bool clip){
    // printf("========== conv_v1::begin conv ==========\n");
    double *img_cuda,*filter_cuda,*output_cuda,*bias_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    int out_width=out_width_;
    

    hipMalloc(&output_cuda,sizeof(double)*filter_num*out_width*out_width);
    filter_cuda = filter;
    bias_cuda=bias;
    if(padding){
        hipMalloc(&img_cuda,sizeof(double)*input_depth*in_width*in_width);
        hipMemset(img_cuda, 0, sizeof(double)*input_depth*in_width*in_width);
        for(int mapid=0;mapid<input_depth;mapid++)
        for(int line=0;line<input_width;line++){
            hipMemcpy(&img_cuda[mapid*in_width*in_width+(padding+line)*in_width+padding], &input[mapid*input_width*input_width+line*input_width], sizeof(double)*input_width, hipMemcpyDeviceToDevice);
        }
    }
    else img_cuda = input;

    int g=(out_width+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 threads(BLOCKSIZE, BLOCKSIZE);
    dim3 grid(g, g, filter_num);

    convKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width, clip);
    if(img_cuda != input)
        hipFree(img_cuda);
    hipFree(input);

    // printf("========== conv_v1::end conv ==========\n");
    return output_cuda;
}

double* conv_group(const int input_depth, const int input_width, 
    const int filter_num, const int out_width_,
    const int filter_width,
    const int padding, const int stride, const int dilation,
    double* filter, double* bias,
    double* input, const bool clip){
    // printf("========== conv_v1::begin conv ==========\n");
    double *img_cuda,*filter_cuda,*output_cuda,*bias_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    int out_width=out_width_;
    

    hipMalloc(&output_cuda,sizeof(double)*filter_num*out_width*out_width);
    filter_cuda = filter;
    bias_cuda=bias;

    if(padding){
        hipMalloc(&img_cuda,sizeof(double)*input_depth*in_width*in_width);
        hipMemset(img_cuda, 0, sizeof(double)*input_depth*in_width*in_width);
        for(int mapid=0;mapid<input_depth;mapid++)
        for(int line=0;line<input_width;line++){
            hipMemcpy(&img_cuda[mapid*in_width*in_width+(padding+line)*in_width+padding], &input[mapid*input_width*input_width+line*input_width], sizeof(double)*input_width, hipMemcpyDeviceToDevice);
        }
    }else img_cuda = input;    

    dim3 threads(1, 1);
    dim3 grid(out_width, out_width, input_depth);

    convGroupKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width, clip);

    if(img_cuda != input)
        hipFree(img_cuda);
    hipFree(input);

    // printf("========== conv_v1::end conv ==========\n");
    return output_cuda;
}



int test_conv1_main()
{
    double* filter=new double[2*3*3*3];
    double* image=new double[3*5*5];
    double* out= new double[2*25];
    double* bias = new double[2];
    for(int i=0;i<2*3*3*3;i++)filter[i]=1;
    for(int i=0;i<3*5*5;i++)image[i]=1;
    bias[0] = 100; bias[1] = 10000;
    double* filter_cu, *image_cu, *out_cu, *bias_cu;
    struct timeval t0;
    gettimeofday(&t0, NULL);
    hipMalloc(&filter_cu, 2*3*3*3*sizeof(double));
    hipMemcpy(filter_cu, filter,2*3*3*3*sizeof(double), hipMemcpyHostToDevice);
    hipMalloc(&image_cu, 3*5*5*sizeof(double));
    hipMemcpy(image_cu, image, 3*5*5*sizeof(double), hipMemcpyHostToDevice);
    hipMalloc(&bias_cu, 2*sizeof(double));
    hipMemcpy(bias_cu, bias,2*sizeof(double),hipMemcpyHostToDevice);
    struct timeval t1, t2;
    gettimeofday(&t1, NULL);
	double timeuse;
    out_cu = conv(3,5,2,5,3,1,1,0,filter_cu,bias_cu,image_cu, false);
    gettimeofday(&t2, NULL);
	timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec) / 1000000.0;
	printf("Conv Use Time:%fs\n", timeuse);
    timeuse = t1.tv_sec - t0.tv_sec + (t1.tv_usec - t0.tv_usec) / 1000000.0;
	printf("Malloc Use Time:%fs\n", timeuse);
    hipMemcpy(out, out_cu, 50*sizeof(double), hipMemcpyDeviceToHost);
    for(int i=0;i<50;i++)printf("%f\n",out[i]);
    return 0;
}
// int main(){test_conv1_main(); return 0;}