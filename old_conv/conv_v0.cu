#include <cstdio>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void convKernel(float *input,float *filter,float *output, float* bias,int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width)
{
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    int input_row = output_row*stride;
    int input_col = output_col*stride;

    for(int fnum=0;fnum<filter_num;fnum++){
        float tmp = 0.0;
        for(int d=0;d<input_depth;d++)
            for(int r=0;r<filter_width;r++)
                for(int c=0;c<filter_width;c++)
                    tmp += input[d*input_width*input_width + (input_row+r)*input_width + input_col+c]
                        *filter[fnum*input_depth*filter_width*filter_width + d*filter_width*filter_width + r*filter_width + c];
        output[fnum*out_width*out_width + output_row*out_width+output_col] = tmp+bias[fnum];
    }
}


void conv(float *input, float *filter, float *output, float* bias, int filter_width, int filter_num,int input_width, int input_depth, int padding, int stride){
    float *img_cuda,*filter_cuda,*output_cuda,*bias_cuda;
    int in_width=input_width+padding*2;
    int out_width=(in_width-filter_width)/stride+1;

    hipMalloc(&img_cuda,sizeof(float)*input_depth*in_width*in_width);
    hipMalloc(&filter_cuda,sizeof(float)*filter_num*input_depth*filter_width*filter_width);
    hipMalloc(&output_cuda,sizeof(float)*filter_num*out_width*out_width);
    hipMalloc(&bias_cuda,sizeof(float)*filter_num);
    hipMemcpy(filter_cuda, filter, sizeof(float)*filter_num*input_depth*filter_width*filter_width, hipMemcpyHostToDevice);
    hipMemcpy(bias_cuda, bias, sizeof(float)*filter_num, hipMemcpyHostToDevice);

    for(int mapid=0;mapid<input_depth;mapid++)
        for(int line=0;line<input_width;line++)
            hipMemcpy(&img_cuda[mapid*in_width*in_width+(padding+line)*in_width+padding], &input[mapid*input_width*input_width+line*input_width], sizeof(float)*input_width, hipMemcpyHostToDevice);


    dim3 threads(1, 1);
    dim3 grid(out_width, out_width);

    convKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width);
    // cudaDeviceSynchronize();

    hipMemcpy(output, output_cuda, sizeof(float)*filter_num*out_width*out_width, hipMemcpyDeviceToHost);

    hipFree(img_cuda);
    hipFree(filter_cuda);
    hipFree(output_cuda);
    hipFree(bias_cuda);
}



int test_conv0_main()
{
     float* filter=new float[2*3*3*3];
     float* image=new float[3*5*5];
     float* out= new float[2*25];
     float* bias = new float[2];
     for(int i=0;i<2*3*3*3;i++)filter[i]=1;
     for(int i=0;i<3*5*5;i++)image[i]=1;
     bias[0] = 100; bias[1] = 10000;
     conv(image,filter,out, bias,3,2,5,3,1,1);
     for(int i=0;i<50;i++)printf("%f\n",out[i]);
     return 0;
}