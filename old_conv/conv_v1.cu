#include "hip/hip_runtime.h"
#include "conv_v1.cuh"
#define BLOCKSIZE 32
using namespace std;

/**
 * the output shape: (out_channel, out_width, out_width)
 * parallel: (out_width, out_width) with maximum #thread per block
 * with fusion of conv and clip
*/

__global__ void convKernel(float *input, float *filter, float *output, float *bias, int filter_width, int filter_num, int input_width, int input_depth, int stride, int out_width, bool clip)
{
    int output_row = blockIdx.x * blockDim.x + threadIdx.x;
    int output_col = blockIdx.y * blockDim.y + threadIdx.y;
    int input_row = output_row * stride;
    int input_col = output_col * stride;
    if (output_row >= out_width || output_col >= out_width)
        return;

    for (int fnum = 0; fnum < filter_num; fnum++)
    {
        float tmp = 0.0;
        for (int d = 0; d < input_depth; d++)
            for (int r = 0; r < filter_width; r++)
                for (int c = 0; c < filter_width; c++)
                    tmp += input[d * input_width * input_width + (input_row + r) * input_width + input_col + c] * filter[fnum * input_depth * filter_width * filter_width + d * filter_width * filter_width + r * filter_width + c];

        tmp += bias[fnum];
        if (clip)
        {
            if (tmp < 0)
                tmp = 0.0;
            else if (tmp > 6)
                tmp = 6.0;
        }
        output[fnum * out_width * out_width + output_row * out_width + output_col] = tmp;
    }
}

/* the float*  are assumed to point to cuda mem. */
float *conv(const int input_depth, const int input_width,
            const int filter_num, const int out_width_,
            const int filter_width,
            const int padding, const int stride, const int dilation,
            float *filter, float *bias,
            float *input, const bool clip)
{
    // printf("========== conv_v1::begin conv ==========\n");
    float *img_cuda, *filter_cuda, *output_cuda, *bias_cuda; //img_cuda for padded tensor.
    int in_width = input_width + padding * 2;
    int out_width = (in_width - filter_width) / stride + 1;

    hipMalloc(&img_cuda, sizeof(float) * input_depth * in_width * in_width);
    // hipMalloc(&filter_cuda,sizeof(float)*filter_num*input_depth*filter_width*filter_width);
    hipMalloc(&output_cuda, sizeof(float) * filter_num * out_width * out_width);
    // hipMalloc(&bias_cuda,sizeof(float)*filter_num);
    // hipMemcpy(filter_cuda, filter, sizeof(float)*filter_num*input_depth*filter_width*filter_width, hipMemcpyHostToDevice);
    filter_cuda = filter;
    // hipMemcpy(bias_cuda, bias, sizeof(float)*filter_num, hipMemcpyHostToDevice);
    bias_cuda = bias;

    for (int mapid = 0; mapid < input_depth; mapid++)
        for (int line = 0; line < input_width; line++)
        {
            hipMemcpy(&img_cuda[mapid * in_width * in_width + (padding + line) * in_width + padding], &input[mapid * input_width * input_width + line * input_width], sizeof(float) * input_width, hipMemcpyDeviceToDevice);
        }

    int g = (out_width + BLOCKSIZE - 1) / BLOCKSIZE;
    dim3 threads(BLOCKSIZE, BLOCKSIZE);
    dim3 grid(g, g);

    convKernel<<<grid, threads>>>(img_cuda, filter_cuda, output_cuda, bias_cuda, filter_width, filter_num, in_width, input_depth, stride, out_width, clip);
    // hipDeviceSynchronize();
    // hipMemcpy(output, output_cuda, sizeof(float)*32*122*122, hipMemcpyDeviceToHost);

    hipFree(img_cuda);
    hipFree(input);
    // hipFree(filter_cuda);
    // hipFree(output_cuda);
    // hipFree(bias_cuda);

    // printf("========== conv_v1::end conv ==========\n");
    return output_cuda;
}

int test_conv1_main()
{
    float *filter = new float[2 * 3 * 3 * 3];
    float *image = new float[3 * 5 * 5];
    float *out = new float[2 * 25];
    float *bias = new float[2];
    for (int i = 0; i < 2 * 3 * 3 * 3; i++)
        filter[i] = 1;
    for (int i = 0; i < 3 * 5 * 5; i++)
        image[i] = 1;
    bias[0] = 100;
    bias[1] = 10000;
    float *filter_cu, *image_cu, *out_cu, *bias_cu;
    hipMalloc(&filter_cu, 2 * 3 * 3 * 3 * sizeof(float));
    hipMemcpy(filter_cu, filter, 2 * 3 * 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&image_cu, 3 * 5 * 5 * sizeof(float));
    hipMemcpy(image_cu, image, 3 * 5 * 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&bias_cu, 2 * sizeof(float));
    hipMemcpy(bias_cu, bias, 2 * sizeof(float), hipMemcpyHostToDevice);

    out_cu = conv(3, 5, 2, 5, 3, 1, 1, 0, filter_cu, bias_cu, image_cu, false);
    hipMemcpy(out, out_cu, 50 * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 50; i++)
        printf("%f\n", out[i]);
    return 0;
}
// int main(){test_conv1_main(); return 0;}