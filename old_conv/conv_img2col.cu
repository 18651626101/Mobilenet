#include "hip/hip_runtime.h"
__global__ void img2col(const int input_depth, const int input_width, const int out_width, const int padding, const int stride, double* input, double* dest){ 
    // 只考虑 kernelsize =  (3,3), stride = 1
    int input_row = blockIdx.x * blockDim.x + threadIdx.x;
    int input_col = blockIdx.y * blockDim.y + threadIdx.y;
    int input_channel = blockIdx.z;
    double tmp = input[((input_channel*input_width)+input_row)*input_width + input_col];
    //每个进程取input中的一个element，把它放到矩阵中该出现的（可能不止一个）的位置上
    // 第一组位置，input的这个element对应卷积核的第三行
    // col: out_width * (input_row-2) + input_col-2
    // row: 9*input_channel +8
    if(input_row >= 2){
        dest[(out_width*(input_row-2)+input_col )+ (9*input_channel+6)*out_width*out_width ] = tmp;
        if(input_col >=1){
            dest[(out_width*(input_row-2)+input_col-1)+ (9*input_channel+7)*out_width*out_width ] = tmp;
            if(input_col>=2)
                dest[(out_width*(input_row-2)+input_col-2)+ (9*input_channel+8)*out_width*out_width ] = tmp;
        }
    }
    // col: out_width * (input_row - 1) + input_col-2
    // row: 9*input_channel + 5
    if(input_row>=1){
        dest[(out_width*(input_row-1)+input_col )+ (9*input_channel+3)*out_width*out_width ] = tmp;
        if(input_col>=1){
            dest[(out_width*(input_row-1)+input_col-1)+ (9*input_channel+4)*out_width*out_width ] = tmp;
            if(input_col>=2)
                dest[(out_width*(input_row-1)+input_col-2)+ (9*input_channel+5)*out_width*out_width ] = tmp;
        }
    }


    dest[(out_width*(input_row)+input_col   )+ (9*input_channel)*out_width*out_width ] = tmp;
    if(input_col >= 1){
        dest[(out_width*(input_row)+input_col-1 )+ (9*input_channel+1)*out_width*out_width ] = tmp;
        if(input_col >= 2)
            dest[(out_width*(input_row)+input_col-2 )+ (9*input_channel+2)*out_width*out_width ] = tmp;
    }

    
}



float* conv_col(const int input_depth, const int input_width,  const int filter_num, const int out_width,    const int filter_width,    const int padding, const int stride, const int dilation,    double* filter, double* bias,    double* input, const bool clip=true){

    //大致流程：先调用img2col函数重排input
    //然后调用经过加入了output padding功能的gemm进行计算
    in_width = input_width;
    


    return NULL;

}