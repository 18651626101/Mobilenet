#include "hip/hip_runtime.h"
#include "gemm.cuh"

__global__ void matMulKernel(const int A_height, const int A_width, const int B_height, const int B_width, double *A, double *B, double *C, double *D)
{
	double Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(row<A_height&&col<B_width){
		for (int i = 0; i < A_width; ++i)
		{
			Cvalue += A[row * A_width + i] * B[i*B_width + col];
		}
		Cvalue += D[row * B_width + col];
		C[row * B_width + col] = Cvalue;
	}
}

void gemm(const int A_height, const int A_width, const int B_height, const int B_width,
			 double *MatrixA, double *MatrixB, double *bias, double *out)
{

	dim3 blockSize(32, 32);
	dim3 gridSize((B_width + blockSize.x - 1) / blockSize.x,
				  (A_height + blockSize.y - 1) / blockSize.y);

	matMulKernel<<<gridSize, blockSize>>>(A_height, A_width, B_height, B_width, MatrixA, MatrixB, out, bias);
}

// int main()
// int test_gemm_main()
// {
// 	double *test_inputd;
// 	double test_input[1 * 2];
// 	double *test_weightd;
// 	double test_weight[3 * 2];
// 	double *test_biasd;
// 	double test_bias[3];
// 	double *test_outputd;
// 	double test_output[3];

// 	for (int i = 0; i < 2; i++)
// 	{
// 		test_input[i] = double(i);
// 		printf("input %f\n", test_input[i]);
// 	}
// 	for (int i = 0; i < 2 * 3; i++)
// 	{
// 		test_weight[i] = double(i);
// 		printf("weight %f\n", test_weight[i]);
// 	}
// 	for (int i = 0; i < 3; i++)
// 	{
// 		test_bias[i] = double(i);
// 		printf("bias %f\n", test_bias[i]);
// 	}

// 	hipMalloc(&test_inputd, sizeof(double) * 2);
// 	hipMemcpy(test_inputd, test_input, sizeof(double) * 2, hipMemcpyHostToDevice);
// 	hipMalloc(&test_weightd, sizeof(double) * 2 * 3);
// 	hipMemcpy(test_weightd, test_weight, sizeof(double) * 2 * 3, hipMemcpyHostToDevice);
// 	hipMalloc(&test_biasd, sizeof(double) * 3);
// 	hipMemcpy(test_biasd, test_bias, sizeof(double) * 3, hipMemcpyHostToDevice);

// 	// test_outputd = gemm(3, 2, 2, 1, test_weightd, test_inputd,test_biasd);

// 	hipMemcpy(test_output, test_outputd, sizeof(double) * 3, hipMemcpyDeviceToHost);
// 	for (int i = 0; i < 3; i++)
// 	{
// 		printf("%f\n", test_output[i]);
// 	}
// 	return 0;
// // }

// int test_gemm_main()
// {
// 	int width = w;
// 	int height = w;

// 	Matrix *A, *B, *C;

// 	hipMallocManaged((void**)&A, sizeof(Matrix));
// 	hipMallocManaged((void**)&B, sizeof(Matrix));
// 	hipMallocManaged((void**)&C, sizeof(Matrix));

// 	int nBytes = width * height * sizeof(double);

// 	hipMallocManaged((void**)&A->elements, nBytes);
// 	hipMallocManaged((void**)&B->elements, nBytes);
// 	hipMallocManaged((void**)&C->elements, nBytes);

// 	A->height = height;
// 	A->width = width;
// 	B->height = height;
// 	B->width = width;
// 	C->height = height;
// 	C->width = width;

// 	for (int i = 0; i < width * height; ++i)
// 	{
// 		A->elements[i] = 1.0;
// 		B->elements[i] = 2.0;
// 	}

// 	dim3 blockSize(32, 32);
// 	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
// 		(height + blockSize.y - 1) / blockSize.y);

// 	struct timeval t1,t2;
// 	gettimeofday(&t1,NULL);
// 	double timeuse;

// 	matMulKernel << < gridSize, blockSize >> >(A, B, C);

// 	hipDeviceSynchronize();

// 	gettimeofday(&t2,NULL);
// 	timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec)/1000000.0;
// 	printf("Use Time:%fs\n", timeuse);

// 	return 0;
// }