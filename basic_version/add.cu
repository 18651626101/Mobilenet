#include "hip/hip_runtime.h"
#include "add.cuh"
#define BLOCKSIZE 32
using namespace std;

__global__ void AddKernel(const int size, double *inputA, double *inputB)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    inputA[k] += inputB[k];

}

void add(const int channel, const int shape, double *inputA, double *inputB)
{
    // printf("========== add:: begin add ==========\n");
    int size = channel*shape*shape;
    int grid_sz = size / BLOCKSIZE;
    if (size % BLOCKSIZE)
        grid_sz++;

    //invoke function on device
    dim3 dimGrid(grid_sz);
    dim3 dimBlock(BLOCKSIZE);
    AddKernel<<<dimGrid, dimBlock>>>(size, (double *)inputA, (double *)inputB);

    //return result
    hipFree(inputB);
    // printf("========== add:: end add ==========\n");
}

// int main()
int test_add_main()
{
    double *test_inputda;
    double test_inputa[3 * 2 * 2];
    double *test_inputdb;
    double test_inputb[3 * 2 * 2];
    // double *test_outputd;
    double test_output[3 * 2 * 2];

    for (int i = 0; i < 12; i++)
    {
        test_inputa[i] = double(i);
        test_inputb[i] = double(i);
        printf("%f %f\n", test_inputa[i], test_inputb[i]);
    }

    hipMalloc(&test_inputda, sizeof(double) * 3 * 2 * 2);
    hipMemcpy(test_inputda, test_inputa, sizeof(double) * 3 * 2 * 2, hipMemcpyHostToDevice);
    hipMalloc(&test_inputdb, sizeof(double) * 3 * 2 * 2);
    hipMemcpy(test_inputdb, test_inputb, sizeof(double) * 3 * 2 * 2, hipMemcpyHostToDevice);   
    
    add(3, 2, test_inputda, test_inputdb);

    hipMemcpy(test_output, test_inputda, sizeof(double) * 3*2*2, hipMemcpyDeviceToHost);
    for (int i = 0; i < 12; i++)
    {
        printf("%f\n", test_output[i]);
    }
    return 0;
}