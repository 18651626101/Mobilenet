#include "hip/hip_runtime.h"
#include "global_avg.cuh"
#define BLOCKSIZE 32
using namespace std;

__global__ void GlobalAvgKernel(double *input, double *output, int depth, int width)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    double amount = 0;
    for (int i = 0; i < width * width; i++)
    {
        amount += input[k * width * width + i];
    }
    output[k] = amount / (width * width);
}

double *global_avg(double *input, int depth, int width)
{
    // printf("========== global_avg:: begin global average pooling ==========\n");
    double *output;
    int grid_sz = depth / BLOCKSIZE;
    if (depth % BLOCKSIZE)
        grid_sz++;

    //alloc memory for output
    hipMalloc(&output, sizeof(double) * depth);

    //invoke function on device
    dim3 dimGrid(grid_sz);
    dim3 dimBlock(BLOCKSIZE);
    GlobalAvgKernel<<<dimGrid, dimBlock>>>((double *)input, (double *)output, depth, width);

    //return result
    hipFree(input);
    // printf("========== global_avg:: end global average pooling ==========\n");
    return output;
}

// int main()
int test_global_main()
{
    double *test_inputd;
    double test_input[3 * 2 * 2];
    double *test_outputd;
    double test_output[3];

    for (int i = 0; i < 12; i++)
    {
        test_input[i] = double(i);
        printf("%f\n", test_input[i]);
    }

    hipMalloc(&test_inputd, sizeof(double) * 3 * 2 * 2);
    hipMemcpy(test_inputd, test_input, sizeof(double) * 3 * 2 * 2, hipMemcpyHostToDevice);

    test_outputd = global_avg(test_inputd, 3, 2);

    hipMemcpy(test_output, test_outputd, sizeof(double) * 3, hipMemcpyDeviceToHost);
    for (int i = 0; i < 3; i++)
    {
        printf("%f\n", test_output[i]);
    }
    return 0;
}