#include "hip/hip_runtime.h"
#include "add_malloc.cuh"
#define BLOCKSIZE 32
using namespace std;

/* AddKernel：加法操作的核函数 */
__global__ void AddKernel(const int size, double *inputA, double *inputB)
{
    //一个thread计算结果矩阵中的一个值
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    inputA[k] += inputB[k];
}

/* 
 * add: 矩阵加法操作
 * params:
 *   channel: 输入的channel个数
 *   shape: 输入的宽度
 *   inputA: 输入，大小为 channel * shape * shape 
 *   inputB: 输入，大小为 channel * shape * shape
 *   output: inputA
 */
void add(const int channel, const int shape, double *inputA, double *inputB)
{
    int size = channel*shape*shape;
    int grid_sz = size / BLOCKSIZE;
    if (size % BLOCKSIZE)
        grid_sz++;

    dim3 dimGrid(grid_sz);
    dim3 dimBlock(BLOCKSIZE);
    AddKernel<<<dimGrid, dimBlock>>>(size, (double *)inputA, (double *)inputB);
}

/* 调试函数 */
int test_add_main()
{
    double *test_inputda;
    double test_inputa[3 * 2 * 2];
    double *test_inputdb;
    double test_inputb[3 * 2 * 2];
    double test_output[3 * 2 * 2];

    for (int i = 0; i < 12; i++)
    {
        test_inputa[i] = double(i);
        test_inputb[i] = double(i);
        printf("%f %f\n", test_inputa[i], test_inputb[i]);
    }

    hipMalloc(&test_inputda, sizeof(double) * 3 * 2 * 2);
    hipMemcpy(test_inputda, test_inputa, sizeof(double) * 3 * 2 * 2, hipMemcpyHostToDevice);
    hipMalloc(&test_inputdb, sizeof(double) * 3 * 2 * 2);
    hipMemcpy(test_inputdb, test_inputb, sizeof(double) * 3 * 2 * 2, hipMemcpyHostToDevice);   
    
    add(3, 2, test_inputda, test_inputdb);

    hipMemcpy(test_output, test_inputda, sizeof(double) * 3*2*2, hipMemcpyDeviceToHost);
    for (int i = 0; i < 12; i++)
    {
        printf("%f\n", test_output[i]);
    }
    return 0;
}