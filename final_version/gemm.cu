#include "hip/hip_runtime.h"
#include "gemm.cuh"
#define BLOCKSIZE 32

/* matMulKernel: 矩阵乘法核函数*/
__global__ void matMulKernel(const int A_height, const int A_width, const int B_height, const int B_width, double *A, double *B, double *C, double *D)
{
	//一个thread计算结果矩阵中的一个值
	double Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(row<A_height&&col<B_width){
		for (int i = 0; i < A_width; ++i)
		{
			Cvalue += A[row * A_width + i] * B[i*B_width + col];
		}
		Cvalue += D[row * B_width + col];
		C[row * B_width + col] = Cvalue;
	}
}

/* 
 * gemm: 通用矩阵乘法操作
 * params:
 *   A_height: 矩阵A的高度
 *   A_width: 矩阵A的高度
 *   B_height: 矩阵B的高度
 *   B_width: 矩阵B的宽度
 *   MatrixA: 矩阵A，大小为 A_height*A_width
 *   MatrixB: 矩阵B，大小为 B_height*B_width
 *   bias: 偏置，大小为 A_height*B_with
 *   out: 输出， 大小为 A_height*B_with
 */
void gemm(const int A_height, const int A_width, const int B_height, const int B_width,
			 double *MatrixA, double *MatrixB, double *bias, double *out)
{

	dim3 blockSize(BLOCKSIZE, BLOCKSIZE);
	dim3 gridSize((B_width + blockSize.x - 1) / blockSize.x,
				  (A_height + blockSize.y - 1) / blockSize.y);

	matMulKernel<<<gridSize, blockSize>>>(A_height, A_width, B_height, B_width, MatrixA, MatrixB, out, bias);
}

/* 调试函数 */
// int test_gemm_main()
// {
// 	double *test_inputd;
// 	double test_input[1 * 2];
// 	double *test_weightd;
// 	double test_weight[3 * 2];
// 	double *test_biasd;
// 	double test_bias[3];
// 	double *test_outputd;
// 	double test_output[3];

// 	for (int i = 0; i < 2; i++)
// 	{
// 		test_input[i] = double(i);
// 		printf("input %f\n", test_input[i]);
// 	}
// 	for (int i = 0; i < 2 * 3; i++)
// 	{
// 		test_weight[i] = double(i);
// 		printf("weight %f\n", test_weight[i]);
// 	}
// 	for (int i = 0; i < 3; i++)
// 	{
// 		test_bias[i] = double(i);
// 		printf("bias %f\n", test_bias[i]);
// 	}

// 	hipMalloc(&test_inputd, sizeof(double) * 2);
// 	hipMemcpy(test_inputd, test_input, sizeof(double) * 2, hipMemcpyHostToDevice);
// 	hipMalloc(&test_weightd, sizeof(double) * 2 * 3);
// 	hipMemcpy(test_weightd, test_weight, sizeof(double) * 2 * 3, hipMemcpyHostToDevice);
// 	hipMalloc(&test_biasd, sizeof(double) * 3);
// 	hipMemcpy(test_biasd, test_bias, sizeof(double) * 3, hipMemcpyHostToDevice);

// 	// test_outputd = gemm(3, 2, 2, 1, test_weightd, test_inputd,test_biasd);

// 	hipMemcpy(test_output, test_outputd, sizeof(double) * 3, hipMemcpyDeviceToHost);
// 	for (int i = 0; i < 3; i++)
// 	{
// 		printf("%f\n", test_output[i]);
// 	}
// 	return 0;
// // }