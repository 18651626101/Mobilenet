#include "hip/hip_runtime.h"
#include "global_avg.cuh"
#define BLOCKSIZE 32
using namespace std;

/* 
 * GlobalAvgKernel: 平均池化操作的卷积核函数
 */
__global__ void GlobalAvgKernel(double *input, double *output, int depth, int width)
{
    // 一个thread计算一个channel对应的值
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    double amount = 0;
    for (int i = 0; i < width * width; i++)
    {
        amount += input[k * width * width + i];
    }
    output[k] = amount / (width * width);
}

/* 
 * global_avg: 平均池化操作
 * params:
 *   input: 输入，维度为 depth * width * width 
 *   depth: 输入的channel个数
 *   width: 输入的宽度
 *   output: 返回值, 维度为 depth * 1 * 1
 */
double *global_avg(double *input, int depth, int width)
{
    double *output;
    int grid_sz = depth / BLOCKSIZE;
    if (depth % BLOCKSIZE)
        grid_sz++;

    hipMalloc(&output, sizeof(double) * depth);

    dim3 dimGrid(grid_sz);
    dim3 dimBlock(BLOCKSIZE);
    GlobalAvgKernel<<<dimGrid, dimBlock>>>((double *)input, (double *)output, depth, width);

    hipFree(input);
    return output;
}

/* 调试函数 */
int test_global_main()
{
    double *test_inputd;
    double test_input[3 * 2 * 2];
    double *test_outputd;
    double test_output[3];

    for (int i = 0; i < 12; i++)
    {
        test_input[i] = double(i);
        printf("%f\n", test_input[i]);
    }

    hipMalloc(&test_inputd, sizeof(double) * 3 * 2 * 2);
    hipMemcpy(test_inputd, test_input, sizeof(double) * 3 * 2 * 2, hipMemcpyHostToDevice);

    test_outputd = global_avg(test_inputd, 3, 2);

    hipMemcpy(test_output, test_outputd, sizeof(double) * 3, hipMemcpyDeviceToHost);
    for (int i = 0; i < 3; i++)
    {
        printf("%f\n", test_output[i]);
    }
    return 0;
}