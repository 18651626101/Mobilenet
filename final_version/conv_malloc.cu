
#include <hip/hip_runtime.h>

#define BLOCKSIZE 8
using namespace std;

/**
 * the output shape: (out_channel, out_width, out_width)
 * parallel: (out_width, out_width) with maximum #thread per block
 * with fusion of conv and clip
*/
__global__ void conv1Kernel(double *input,double *filter,double *output, double* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, int next_padding, bool clip){
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    // int input_row = output_row*stride;
    // int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;
    out_width += next_padding * 2;

    int fnum = blockIdx.z;
    double tmp = 0.0;
    // for(int r=0;r<filter_width;r++)
    // for(int c=0;c<filter_width;c++)
    for(int d=0;d<input_depth;d++)
        tmp = tmp + input[d * input_width*input_width + (output_row)*input_width + output_col]*filter[fnum*input_depth + d];
    // __syncthreads();
    // if(threadIdx.z==0) {
        tmp += bias[fnum];
    //     __syncthreads();
    // }
    if(clip){
        if(tmp < 0) tmp = 0.0;
        else if(tmp > 6) tmp = 6.0;
    }
    output[fnum*out_width*out_width + (output_row+next_padding)*out_width+output_col+next_padding] = tmp;
}

__global__ void convKernel(double *input,double *filter,double *output, double* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, int next_padding, bool clip){
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    int input_row = output_row*stride;
    int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;
    out_width += next_padding * 2;

    int fnum = blockIdx.z;
    double tmp = 0.0;
    for(int d=0;d<input_depth;d++)
    for(int r=0;r<filter_width;r++)
    for(int c=0;c<filter_width;c++)
        tmp += input[d*input_width*input_width + (input_row+r)*input_width + input_col+c]*filter[fnum*input_depth*filter_width*filter_width + d*filter_width*filter_width + r*filter_width + c];
    
    tmp += bias[fnum];
    if(clip){
        if(tmp < 0) tmp = 0.0;
        else if(tmp > 6) tmp = 6.0;
    }
    output[fnum*out_width*out_width + (output_row+next_padding)*out_width+output_col+next_padding] = tmp;
}

__global__ void conv1GroupKernel(double *input,double *filter,double *output, double* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, int next_padding, bool clip){
    int layer_idx = blockIdx.z;
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    // int input_row = output_row*stride;
    // int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;
    out_width += next_padding * 2;

    
        double tmp = 0.0;
        // for(int r=0;r<filter_width;r++)
        // for(int c=0;c<filter_width;c++){
            tmp = input[layer_idx*input_width*input_width + (output_row)*input_width + output_col]*filter[layer_idx];
        // }
        
        tmp += bias[layer_idx];
        if(clip){
            if(tmp < 0) tmp = 0.0;
            else if(tmp > 6) tmp = 6.0;
        }
        output[layer_idx*out_width*out_width + (output_row+next_padding)*out_width+output_col+next_padding] = tmp;
    
}
__global__ void convGroupKernel(double *input,double *filter,double *output, double* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, int next_padding, bool clip){
    int layer_idx = blockIdx.z;
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    int input_row = output_row*stride;
    int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;
    out_width += next_padding * 2;

    
        double tmp = 0.0;
        for(int r=0;r<filter_width;r++)
        for(int c=0;c<filter_width;c++){
            tmp += input[layer_idx*input_width*input_width + (input_row+r)*input_width + input_col+c]*filter[layer_idx*filter_width*filter_width + r*filter_width + c];
        }
        
        tmp += bias[layer_idx];
        if(clip){
            if(tmp < 0) tmp = 0.0;
            else if(tmp > 6) tmp = 6.0;
        }
        output[layer_idx*out_width*out_width + (output_row+next_padding)*out_width+output_col+next_padding] = tmp;
    
}


/* the double*  are assumed to point to cuda mem. */
void conv(const int input_depth, const int input_width, 
    const int filter_num, const int out_width_,
    const int filter_width,
    const int padding, const int stride, const int dilation,
    double* filter, double* bias,
    double* &input, double* &output, int next_padding, const bool clip = true){
    // printf("========== conv_malloc::begin conv ==========\n");
    double *img_cuda,*filter_cuda,*output_cuda,*bias_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    int out_width=out_width_+next_padding*2;
    size_t outsize = sizeof(double)*filter_num*out_width*out_width;
    

    // cudaMalloc(&output_cuda,outsize);
    output_cuda = output;
    hipMemset(output_cuda, 0, outsize);
    filter_cuda = filter;
    bias_cuda=bias;
    img_cuda = input;

    int g=(out_width_+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 grid(g, g, filter_num);
    dim3 threads(BLOCKSIZE, BLOCKSIZE);  
    if(filter_width == 1){  
        conv1Kernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width_, next_padding, clip);
        }
    else {
        dim3 threads(BLOCKSIZE, BLOCKSIZE);
        convKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width_, next_padding, clip);
    }
    
    input = output_cuda;
    // printf("========== conv_v1::end conv ==========\n");
    
}

void conv_group(const int input_depth, const int input_width, 
    const int filter_num, const int out_width_,
    const int filter_width,
    const int padding, const int stride, const int dilation,
    double* filter, double* bias,
    double* &input, double* &output,int next_padding, const bool clip = true){
    // printf("========== conv_v1::begin conv ==========\n");
    double *img_cuda,*filter_cuda,*output_cuda,*bias_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    int out_width=out_width_+next_padding*2;
    size_t outsize = sizeof(double)*filter_num*out_width*out_width;

    // cudaMalloc(&output_cuda,outsize);
    output_cuda = output;
    hipMemset(output_cuda,0, outsize);
    filter_cuda = filter;
    bias_cuda=bias;

    img_cuda = input;    

    // dim3 threads(1, 1);
    // dim3 grid(out_width_, out_width_, input_depth);
    int g=(out_width_+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 threads(BLOCKSIZE, BLOCKSIZE);
    dim3 grid(g, g, filter_num);
    // if(filter_width == 1)
    //     conv1GroupKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width_,next_padding,clip);
    // else 
        convGroupKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width_,next_padding,clip);

    input = output;

    // printf("========== conv_v1::end conv ==========\n");
    
}

double* pad(const int input_depth, const int input_width, const int padding, double* input){
    double *img_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    hipMalloc(&img_cuda,sizeof(double)*input_depth*in_width*in_width);
    hipMemset(img_cuda, 0, sizeof(double)*input_depth*in_width*in_width);
    for(int mapid=0;mapid<input_depth;mapid++)
    for(int line=0;line<input_width;line++){
        hipMemcpy(&img_cuda[mapid*in_width*in_width+(padding+line)*in_width+padding], &input[mapid*input_width*input_width+line*input_width], sizeof(double)*input_width, hipMemcpyHostToDevice);
    }
    return img_cuda;
}

// int test_conv1_main()
// {
//     double* filter=new double[2*3*3*3];
//     double* image=new double[3*5*5];
//     double* out= new double[2*25];
//     double* bias = new double[2];
//     for(int i=0;i<2*3*3*3;i++)filter[i]=1;
//     for(int i=0;i<3*5*5;i++)image[i]=1;
//     bias[0] = 100; bias[1] = 10000;
//     double* filter_cu, *image_cu, *out_cu, *bias_cu;
    
//     cudaMalloc(&filter_cu, 2*3*3*3*sizeof(double));
//     cudaMemcpy(filter_cu, filter,2*3*3*3*sizeof(double), cudaMemcpyHostToDevice);
//     cudaMalloc(&image_cu, 3*5*5*sizeof(double));
//     cudaMemcpy(image_cu, image, 3*5*5*sizeof(double), cudaMemcpyHostToDevice);
//     cudaMalloc(&bias_cu, 2*sizeof(double));
//     cudaMemcpy(bias_cu, bias,2*sizeof(double),cudaMemcpyHostToDevice);
    
	
//     conv(3,5,2,5,3,1,1,0,filter_cu,bias_cu,image_cu,out_cu,0,false);

//     cudaMemcpy(out, out_cu, 50*sizeof(double), cudaMemcpyDeviceToHost);
//     for(int i=0;i<50;i++)printf("%f\n",out[i]);
//     return 0;
// }
// int main(){test_conv1_main(); return 0;}