
#include <hip/hip_runtime.h>

#define BLOCKSIZE 8
using namespace std;

/**
 * conv1Kernel只处理kernel size 1*1的卷积，省去了许多不必要的循环、乘法等操作。
 * 原始版本（及没有去掉循环的版本）可以参照convKernel
*/
__global__ void conv1Kernel(double *input,double *filter,double *output, double* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, int next_padding, bool clip){
    /* params:
     * input_width: 已经考虑padding在内的input宽度
     * out_width: 未考虑next padding的情况下的输出整体图像的宽度
     * next_padding: 需要输入下一层的padding参数，在本层产生运算结果的同时进行pad
     */
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    if(output_row >= out_width || output_col >= out_width) return;
    out_width += next_padding * 2; //此后out_width表示加pad之后的宽度

    int fnum = blockIdx.z;
    double tmp = 0.0;
    for(int d=0;d<input_depth;d++)
        tmp = tmp + input[d * input_width*input_width + (output_row)*input_width + output_col]*filter[fnum*input_depth + d];
    tmp += bias[fnum];
    if(clip){
        if(tmp < 0) tmp = 0.0;
        else if(tmp > 6) tmp = 6.0;
    }
    output[fnum*out_width*out_width + (output_row+next_padding)*out_width+output_col+next_padding] = tmp;
}


__global__ void convKernel(double *input,double *filter,double *output, double* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, int next_padding, bool clip){
    /* 
     * 每个线程负责计算output中的一个double值
     * params:
     * input_width: 已经考虑padding在内的input宽度
     * out_width: 未考虑next padding的情况下的输出整体图像的宽度
     * next_padding: 需要输入下一层的padding参数，在本层产生运算结果的同时进行pad
     */
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    int input_row = output_row*stride;
    int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;
    out_width += next_padding * 2;//此后out_width表示加pad之后的宽度

    int fnum = blockIdx.z;
    double tmp = 0.0;
    for(int d=0;d<input_depth;d++)
    for(int r=0;r<filter_width;r++)
    for(int c=0;c<filter_width;c++)
        tmp += input[d*input_width*input_width + (input_row+r)*input_width + input_col+c]*filter[fnum*input_depth*filter_width*filter_width + d*filter_width*filter_width + r*filter_width + c];
    
    tmp += bias[fnum];
    if(clip){
        if(tmp < 0) tmp = 0.0;
        else if(tmp > 6) tmp = 6.0;
    }
    output[fnum*out_width*out_width + (output_row+next_padding)*out_width+output_col+next_padding] = tmp;
}

__global__ void convGroupKernel(double *input,double *filter,double *output, double* bias, int filter_width,int filter_num,int input_width,int input_depth,int stride,int out_width, int next_padding, bool clip){
    int layer_idx = blockIdx.z;
    int output_row = blockIdx.x*blockDim.x+threadIdx.x;
    int output_col = blockIdx.y*blockDim.y+threadIdx.y;
    int input_row = output_row*stride;
    int input_col = output_col*stride;
    if(output_row >= out_width || output_col >= out_width) return;
    out_width += next_padding * 2;

    
    double tmp = 0.0; //本地存储，减少global mem的访问次数
    //二重循环计算9个元素与9个元素对应相乘再相加
    for(int r=0;r<filter_width;r++)
    for(int c=0;c<filter_width;c++){
        tmp += input[layer_idx*input_width*input_width + (input_row+r)*input_width + input_col+c]*filter[layer_idx*filter_width*filter_width + r*filter_width + c];
    }
    
    tmp += bias[layer_idx];
    if(clip){
        if(tmp < 0) tmp = 0.0;
        else if(tmp > 6) tmp = 6.0;
    }
    output[layer_idx*out_width*out_width + (output_row+next_padding)*out_width+output_col+next_padding] = tmp;
    
}



void conv(const int input_depth, const int input_width, 
    const int filter_num, const int out_width_,
    const int filter_width,
    const int padding, const int stride, const int dilation,
    double* filter, double* bias,
    double* &input, double* &output, int next_padding, const bool clip = true){
    /* 
     * 此处假设filter, bias, input均指向global mem，而非host mem。
     * 整个网络的计算、中间结果的存储，都在gpu上进行。
     */
    double *img_cuda,*filter_cuda,*output_cuda,*bias_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    int out_width=out_width_+next_padding*2;
    size_t outsize = sizeof(double)*filter_num*out_width*out_width;
    
    output_cuda = output;
    hipMemset(output_cuda, 0, outsize);
    filter_cuda = filter;
    bias_cuda=bias;
    img_cuda = input;

    int g=(out_width_+BLOCKSIZE-1)/BLOCKSIZE; //向上取整
    dim3 grid(g, g, filter_num);
    dim3 threads(BLOCKSIZE, BLOCKSIZE);  
    if(filter_width == 1){  
        conv1Kernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width_, next_padding, clip);
        }
    else {
        dim3 threads(BLOCKSIZE, BLOCKSIZE);
        convKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width_, next_padding, clip);
    }
    
    input = output_cuda;    
}

void conv_group(const int input_depth, const int input_width, 
    const int filter_num, const int out_width_,
    const int filter_width,
    const int padding, const int stride, const int dilation,
    double* filter, double* bias,
    double* &input, double* &output,int next_padding, const bool clip = true){
    // 本函数处理group参数等于input_depth的卷积计算，结构与conv几乎完全相同
    double *img_cuda,*filter_cuda,*output_cuda,*bias_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    int out_width=out_width_+next_padding*2;
    size_t outsize = sizeof(double)*filter_num*out_width*out_width;

    output_cuda = output;
    hipMemset(output_cuda,0, outsize);
    filter_cuda = filter;
    bias_cuda=bias;
    img_cuda = input;    

    int g=(out_width_+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 threads(BLOCKSIZE, BLOCKSIZE);
    dim3 grid(g, g, filter_num);
    convGroupKernel<<<grid,threads>>>(img_cuda,filter_cuda,output_cuda,bias_cuda,filter_width,filter_num,in_width,input_depth,stride,out_width_,next_padding,clip);

    input = output;    
}

double* pad(const int input_depth, const int input_width, const int padding, double* input){
    double *img_cuda; //img_cuda for padded tensor.
    int in_width=input_width+padding*2;
    hipMalloc(&img_cuda,sizeof(double)*input_depth*in_width*in_width);
    hipMemset(img_cuda, 0, sizeof(double)*input_depth*in_width*in_width);
    for(int mapid=0;mapid<input_depth;mapid++)
    for(int line=0;line<input_width;line++){
        hipMemcpy(&img_cuda[mapid*in_width*in_width+(padding+line)*in_width+padding], &input[mapid*input_width*input_width+line*input_width], sizeof(double)*input_width, hipMemcpyHostToDevice);
    }
    return img_cuda;
}


// 以下是开发过程中为了检验conv函数正确性而设计的测试函数
// int test_conv1_main()
// {
//     double* filter=new double[2*3*3*3];
//     double* image=new double[3*5*5];
//     double* out= new double[2*25];
//     double* bias = new double[2];
//     for(int i=0;i<2*3*3*3;i++)filter[i]=1;
//     for(int i=0;i<3*5*5;i++)image[i]=1;
//     bias[0] = 100; bias[1] = 10000;
//     double* filter_cu, *image_cu, *out_cu, *bias_cu;
    
//     cudaMalloc(&filter_cu, 2*3*3*3*sizeof(double));
//     cudaMemcpy(filter_cu, filter,2*3*3*3*sizeof(double), cudaMemcpyHostToDevice);
//     cudaMalloc(&image_cu, 3*5*5*sizeof(double));
//     cudaMemcpy(image_cu, image, 3*5*5*sizeof(double), cudaMemcpyHostToDevice);
//     cudaMalloc(&bias_cu, 2*sizeof(double));
//     cudaMemcpy(bias_cu, bias,2*sizeof(double),cudaMemcpyHostToDevice);
    
	
//     conv(3,5,2,5,3,1,1,0,filter_cu,bias_cu,image_cu,out_cu,0,false);

//     cudaMemcpy(out, out_cu, 50*sizeof(double), cudaMemcpyDeviceToHost);
//     for(int i=0;i<50;i++)printf("%f\n",out[i]);
//     return 0;
// }
// int main(){test_conv1_main(); return 0;}